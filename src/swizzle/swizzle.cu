#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>

#include "matrix.h"

template <int TILE_Y = 32, int TILE_X = 32>
__global__ void swizzle_transpose(float *A, int rows, int cols, float *B) {
  __shared__ float tile[TILE_Y][TILE_X];
  int r = blockDim.y * blockIdx.y + threadIdx.y;
  int c = blockDim.x * blockIdx.x + threadIdx.x;
  int A_linear_idx = r * cols + c;

  int swizzle_idx = (r ^ c) % TILE_X; // mouldo columns to ensure valid indices

  tile[r][swizzle_idx] = A[A_linear_idx];

  __syncthreads();
  int r_out = blockDim.x * blockIdx.x + threadIdx.y;
  int c_out = blockDim.y * blockIdx.y + threadIdx.x;
  int B_linear_idx = c_out * rows + r_out;

  B[B_linear_idx] = tile[r][swizzle_idx];
}

int div_up(int a, int b) { return (a + b - 1) / b; }

int main() {
  constexpr int rows = 64;
  constexpr int cols = 32;
  constexpr int size = rows * cols;
  constexpr int size_bytes = size * sizeof(float);

  constexpr int TILE_X = 32;
  constexpr int TILE_Y = 32;

  float A[size];
  float B[size];

  initialize_matrix(A, rows, cols);
  std::cout << "Matrix A: " << std::endl;
  print_matrix(A, rows, cols);

  float *A_device, *B_device;

  hipMalloc((void **)&A_device, size_bytes);
  hipMalloc((void **)&B_device, size_bytes);

  hipMemcpy(A_device, A, size_bytes, hipMemcpyHostToDevice);

  dim3 threadsPerBlock(TILE_Y, TILE_X);
  dim3 blocksPerGrid(div_up(rows, threadsPerBlock.y),
                     div_up(cols, threadsPerBlock.x));

  swizzle_transpose<TILE_Y, TILE_X>
      <<<blocksPerGrid, threadsPerBlock>>>(A_device, rows, cols, B_device);

  hipMemcpy(B, B_device, size_bytes, hipMemcpyDeviceToHost);

  std::cout << "Matrix B: " << std::endl;
  print_swizzle_matrix(B, cols, rows);

  hipFree(A_device);
  hipFree(B_device);

  return 0;
}