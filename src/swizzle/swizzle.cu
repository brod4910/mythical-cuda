#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>

#include "matrix.h"

int div_up(int a, int b) { return (a + b - 1) / b; }

template <int TILE_Y = 32, int TILE_X = 32>
__global__ void swizzle_transpose(float *A, int rows, int cols, float *B) {
  __shared__ float tile[TILE_Y][TILE_X];
  int r = blockDim.y * blockIdx.y + threadIdx.y; // row = (32 * by) + ty
  int c = blockDim.x * blockIdx.x + threadIdx.x; // col = (32 * bx) + tx
  int A_linear_idx = r * cols + c;               // linear 1-D index

  int swizzle_idx =
      (r ^ c) %
      TILE_X; // mouldo swizzle index by TILE_X to ensure valid indices
  int tile_y = r % TILE_Y; // modulo rows by TILE_Y to ensure valid indices

  tile[tile_y][swizzle_idx] =
      A[A_linear_idx]; // bank-conflict free swizzled store

  __syncthreads();

  // TODO: Fix indices of destination, currently transposing the tile 
  // and not the whole matrix
  int r_out = blockDim.x * blockIdx.x + threadIdx.y; // row = (32 * bx) + ty
  int c_out = blockDim.y * blockIdx.y + threadIdx.x; // col = (32 * by) + tx
  int B_linear_idx = c_out * rows + r_out; // tranpose linear 1-D index

  B[B_linear_idx] =
      tile[tile_y][swizzle_idx]; // bank-conflict free swizzled load
}

template <typename T, int TILE_Y = 32, int TILE_X = 32>
__global__ void
swizzle_bilinear_interpolation(const T *__restrict__ A, int input_rows,
                               int input_cols, T *__restrict__ B, int out_rows,
                               int out_cols, float scale_y, float scale_x) {
  __shared__ T ATile[TILE_Y][TILE_X];
  // __shared__ BTile[32][32];

  int r = blockDim.y * blockIdx.y + threadIdx.y;
  int c = blockDim.x * blockIdx.x + threadIdx.x;

  int swizzle_idx =
      (r ^ c) %
      TILE_X; // mouldo swizzle index by TILE_X to ensure valid indices
  // int tile_x = c % TILE_X;
  int tile_y = r % TILE_Y; // modulo rows by TILE_Y to ensure valid indices

  // ATile[tile_y][tile_x] = __ldg(&A[r * input_cols + c]); // possibly remove ldg
  ATile[tile_y][swizzle_idx] = A[r * input_cols + c];

  __syncthreads();

  float x = c * scale_x;
  float y = r * scale_y;

  float x_floor = floor(x);
  float y_floor = floor(y);
  float x_ceil = x_floor + 1;
  float y_ceil = y_floor + 1;

  float w_floor_x = x - x_floor;
  float w_ceil_x = x_ceil - x;

  float w_floor_y = y - y_floor;
  float w_ceil_y = y_ceil - y;

  int x1 = ((int)x_floor ^ c) % TILE_X;
  int y1 = (int)y_floor % TILE_Y;
  int x2 = ((int)x_ceil ^ c) % TILE_X;
  int y2 = (int)y_ceil % TILE_Y;


  T ap = ATile[y1][x1];
  T bp = ATile[y1][x2];
  T cp = ATile[y2][x1];
  T dp = ATile[y2][x2];

  // The Parallel Method to Reduce Individual Elements Products into a Final Sum???
  T pixel = (ap * (w_ceil_x * w_ceil_y)) + (bp * (w_ceil_x * w_floor_y)) + (cp * (w_floor_x * w_ceil_y)) + (dp * (w_floor_x * w_floor_y));
  int B_linear_idx = r * out_cols + c;

  B[B_linear_idx] = pixel;
}

template <int Rows, int Cols, int TILE_Y = 32, int TILE_X = 32>
void launch_swizzle_transpose() {
  constexpr int size = Rows * Cols;
  constexpr int size_bytes = size * sizeof(float);

  float A[size];
  float B[size];

  initialize_matrix(A, Rows, Cols);
  std::cout << "Matrix A: " << std::endl;
  print_matrix<float, TILE_Y>(A, Rows, Cols);

  float *A_device, *B_device;

  hipMalloc((void **)&A_device, size_bytes);
  hipMalloc((void **)&B_device, size_bytes);

  hipMemcpy(A_device, A, size_bytes, hipMemcpyHostToDevice);

  dim3 threadsPerBlock(TILE_Y, TILE_X);
  dim3 blocksPerGrid(div_up(Rows, threadsPerBlock.y),
                     div_up(Cols, threadsPerBlock.x));

  swizzle_transpose<TILE_Y, TILE_X>
      <<<blocksPerGrid, threadsPerBlock>>>(A_device, Rows, Cols, B_device);

  hipMemcpy(B, B_device, size_bytes, hipMemcpyDeviceToHost);

  std::cout << "Matrix B: " << std::endl;
  print_swizzle_matrix<float, TILE_X>(B, Cols, Rows);

  hipFree(A_device);
  hipFree(B_device);
}

template <typename T, int Rows, int Cols, int Out_Rows, int Out_Cols,
          int TILE_Y = 32, int TILE_X = 32>
void launch_swizzle_bilinear() {
  constexpr int size = Rows * Cols;
  constexpr int size_bytes = size * sizeof(T);

  constexpr int out_size = Out_Rows * Out_Cols;
  constexpr int out_size_bytes = out_size * sizeof(T);

  T A[size];
  T B[out_size];

  initialize_matrix(A, Rows, Cols);
  std::cout << "Matrix A: " << std::endl;
  // print_matrix<T, TILE_Y>(A, Rows, Cols);

  T *A_device, *B_device;

  hipMalloc((void **)&A_device, size_bytes);
  hipMalloc((void **)&B_device, out_size_bytes);

  hipMemcpy(A_device, A, size_bytes, hipMemcpyHostToDevice);

  dim3 threadsPerBlock(TILE_Y, TILE_X);
  dim3 blocksPerGrid(div_up(Out_Rows, threadsPerBlock.y),
                     div_up(Out_Cols, threadsPerBlock.x));

  constexpr float scale_x = (Cols - 1.) / (Out_Cols - 1.);
  constexpr float scale_y = (Rows - 1.) / (Out_Rows - 1.);

  swizzle_bilinear_interpolation<T, TILE_Y, TILE_X>
      <<<blocksPerGrid, threadsPerBlock>>>(
          A_device, Rows, Cols, B_device, Out_Rows, Out_Cols, scale_y, scale_x);

  hipMemcpy(B, B_device, out_size_bytes, hipMemcpyDeviceToHost);

  std::cout << "Matrix B: " << std::endl;
  print_swizzle_matrix<T, TILE_X>(B, Out_Rows, Out_Cols);

  hipFree(A_device);
  hipFree(B_device);
}

int main() {
  constexpr int rows = 128;
  constexpr int cols = 128;

  constexpr int out_rows = 1024;
  constexpr int out_cols = 1024;

  constexpr int TILE_X = 32;
  constexpr int TILE_Y = 32;

  launch_swizzle_transpose<rows, cols, TILE_Y, TILE_X>();
  launch_swizzle_bilinear<float, rows, cols, out_rows, out_cols, TILE_Y,
                          TILE_X>();
  return 0;
}