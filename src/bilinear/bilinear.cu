#include "hip/hip_runtime.h"
#include "matrix.h"
#include "cuutil.h"

template <typename T, int TILE_X = 32, int TILE_Y = 32>
__global__ void
bilinear_interpolation_v1(const T *__restrict__ A, int input_rows,
                               int input_cols, T *__restrict__ B, int out_rows,
                               int out_cols, float scale_y, float scale_x) {
  int r = blockDim.y * blockIdx.y + threadIdx.y;
  int c = blockDim.x * blockIdx.x + threadIdx.x;

  float x = c * scale_x;
  float y = r * scale_y;

  int x_floor = floor(x);
  int y_floor = floor(y);
  int x_ceil = x_floor + 1;
  int y_ceil = y_floor + 1;

  float d_x = x - x_floor;
  float d_y = y - y_floor;

  // The Parallel Method to Reduce Individual Elements Products into a Final Sum???
  float w00 = (1 - d_x) * (1 - d_y);
  float w01 = d_x * (1 - d_y);
  float w10 = (1 - d_x) * d_y;
  float w11 = d_x * d_y;

  T ap = A[y_floor * input_cols + x_floor];
  T bp = A[y_floor * input_cols + x_ceil];;
  T cp = A[y_ceil * input_cols + x_floor];;
  T dp = A[y_ceil * input_cols + x_ceil];;

  T q00 = ap * w00;
  T q01 = bp * w01;
  T q10 = cp * w10;
  T q11 = dp * w11;

  T pixel = q00 + q01 + q10 + q11;
  B[r * out_cols + c] = pixel;
}

template <typename T, int TILE_X = 32, int TILE_Y = 32>
__global__ void
bilinear_interpolation_v2(const T *__restrict__ A, int input_rows,
                               int input_cols, T *__restrict__ B, int out_rows,
                               int out_cols, float scale_y, float scale_x) {
  __shared__ T tile[TILE_Y][TILE_X];

  int r = blockDim.y * blockIdx.y + threadIdx.y;
  int c = blockDim.x * blockIdx.x + threadIdx.x;
  
  float in_scale_x = (out_cols - 1.f) / (input_cols - 1.f); // 8 out pixels in x use 1 input pixel
  float in_scale_y = (out_rows - 1.f) / (input_rows - 1.f); // 8 out pixels in y use 1 input pixel 
  
  int tile_y = r % TILE_Y;
  int tile_x = c % TILE_X;

  tile[tile_y][tile_x] = __ldg(&A[r * input_cols + c]);

  __syncthreads();

  // int x_floor = c;
  // int y_floor = r;
  // int x_ceil = x_floor + 1;
  // int y_ceil = y_floor + 1;

  int out_y = in_scale_y * r;
  int out_x = in_scale_x * c;

  int num_y = in_scale_y;
  int num_x = in_scale_x;

  for (int i = 0; i < num_y; ++i) {
    int sy = (out_y + num_y * i) + threadIdx.y;
    float y = sy * scale_y;

    if (sy >= out_rows) {
      return;
    }

    for (int j = 0; j < num_x; ++j) {
      int sx = (out_x + num_x * j) + threadIdx.x;

      if (sx >= out_cols) {
        return;
      }

      float x = sx * scale_x;

      float d_x = x - c;
      float d_y = y - r;

      float w00 = (1 - d_x) * (1 - d_y);
      float w01 = d_x * (1 - d_y);
      float w10 = (1 - d_x) * d_y;
      float w11 = d_x * d_y;

      int tfx = (int)floor(x) % TILE_X;
      int tfy = (int)floor(y) % TILE_Y;

      T ap = tile[tfy][tfx];
      T bp = tile[tfy][tfx + 1];
      T cp = tile[tfy + 1][tfx];
      T dp = tile[tfy + 1][tfx + 1];

      T q00 = ap * w00;
      T q01 = bp * w01;
      T q10 = cp * w10;
      T q11 = dp * w11;

      T pixel = q00 + q01 + q10 + q11;
      B[sy * out_cols + sx] = pixel;
    }
  }
}

template <typename T, int Rows, int Cols, int Out_Rows, int Out_Cols,
          int TILE_X = 32, int TILE_Y = 32>
void launch_swizzle_bilinear() {
  constexpr int size = Rows * Cols;
  constexpr int size_bytes = size * sizeof(T);

  constexpr int out_size = Out_Rows * Out_Cols;
  constexpr int out_size_bytes = out_size * sizeof(T);

  T A[size];
  T B[out_size];

  initialize_matrix(A, Rows, Cols);
  std::cout << "Matrix A: " << std::endl;
  print_matrix<T, TILE_Y>(A, Rows, Cols);

  T *A_device, *B_device;

  hipMalloc((void **)&A_device, size_bytes);
  hipMalloc((void **)&B_device, out_size_bytes);

  hipMemcpy(A_device, A, size_bytes, hipMemcpyHostToDevice);

  // dim3 threadsPerBlock(TILE_X, TILE_Y);
  // dim3 blocksPerGrid(div_up(Out_Rows, threadsPerBlock.y),
  //                    div_up(Out_Cols, threadsPerBlock.x));

  constexpr float scale_x = (Cols - 1.) / (Out_Cols - 1.);
  constexpr float scale_y = (Rows - 1.) / (Out_Rows - 1.);
  
  dim3 threadsPerBlock(TILE_X, TILE_Y);
  dim3 blocksPerGrid(div_up(Cols, threadsPerBlock.x), 
                     div_up(Rows, threadsPerBlock.y));

  bilinear_interpolation_v2<T, TILE_X, TILE_Y>
      <<<blocksPerGrid, threadsPerBlock>>>(
          A_device, Rows, Cols, B_device, Out_Rows, Out_Cols, scale_y, scale_x);

  hipMemcpy(B, B_device, out_size_bytes, hipMemcpyDeviceToHost);

  std::cout << "Matrix B: " << std::endl;
  print_swizzle_matrix<T, TILE_X>(B, Out_Rows, Out_Cols);

  hipFree(A_device);
  hipFree(B_device);
}

int main() {
  constexpr int rows = 128;
  constexpr int cols = 128;

  constexpr int out_rows = 1024;
  constexpr int out_cols = 1024;

  constexpr int TILE_X = 4;
  constexpr int TILE_Y = 4;

  launch_swizzle_bilinear<float, rows, cols, out_rows, out_cols, TILE_X,
                          TILE_Y>();
  return 0;
}